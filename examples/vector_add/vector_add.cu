
#include <hip/hip_runtime.h>
template <typename T, int block_size_x>
__global__ void vector_add(T* c,  const T* a,  const T* b, int n) {
    int i = blockIdx.x * block_size_x + threadIdx.x;
    if (i<n) {
        c[i] = a[i] + b[i];
    }
}
